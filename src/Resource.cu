#include "hip/hip_runtime.h"
#include "Resource.cuh"
#include "GLUtils.h"
#include <glad/glad.h>
#include <cuda_gl_interop.h>
// #include <GLFW/glfw3.h>
#include "Global.h"

ImageResource::ImageResource(int width, int height)
{
    init_flag = false;

    pbo = generatePixelBufferObject(width, height, 3);
    // printf("pbo id: %d\n", pbo);
    auto err = hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);
    if (err != hipSuccess)
    {
        printf("CUDA error in registering buffer: %s\n", hipGetErrorString(err));
    }
    pixel_size = width * height;
    // create frame_buffer in device
    // frame_buffer = new unsigned char[3 * scene->get_pixels()];
    err = hipMalloc((void**)&device_frame_buffer, sizeof(uchar3) * pixel_size);
    if (err != hipSuccess)
    {
        printf("CUDA error in allocating memory: %s\n", hipGetErrorString(err));
        throw std::runtime_error("Render.device_frame_buffer to cuda.");
    }

    init_flag = true;
}

ImageResource::~ImageResource()
{
    hipFree(device_frame_buffer);
    hipGraphicsUnregisterResource(cuda_pbo_resource);
    glDeleteBuffers(1, &pbo);
}

void ImageResource::loadImageFromDevice()
{
    // write device_frame_buffer to pbo
            
    auto err = hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
    if (err != hipSuccess)
    {
        printf("CUDA error in mapping resources: %s\n", hipGetErrorString(err));
    }
    size_t num_bytes = sizeof(uchar3) * pixel_size; 
    uchar3* cuda_pbo_pointer;
    err = hipGraphicsResourceGetMappedPointer((void **)&cuda_pbo_pointer, &num_bytes, cuda_pbo_resource);
    if (err != hipSuccess)
    {
        printf("CUDA error in getting mapped pointer: %s\n", hipGetErrorString(err));
    }
    err = hipMemcpy((void *)cuda_pbo_pointer, (void *)device_frame_buffer, sizeof(uchar3) * pixel_size, hipMemcpyDeviceToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA error in memory copy: %s\n", hipGetErrorString(err));
    }
    err = hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
    if (err != hipSuccess)
    {
         printf("CUDA error in umapping resources: %s\n", hipGetErrorString(err));
    }
}

ImagePool::ImagePool(int width, int height)
{
    image_pool = std::vector<ImageResource*>(QUEUE_SIZE);

    for (int i = 0; i < QUEUE_SIZE; i++)
    {
        ImageResource* ir = new ImageResource(width, height);
        image_pool[i] = ir;
    }

    cur = 0;
}

ImagePool::~ImagePool()
{
    for (int i = 0; i < QUEUE_SIZE; i++)
    {
        delete image_pool[i];
    }
    image_pool.clear();
}

ImageResource* ImagePool::get(int idx) 
{
    if (image_pool.size() > 0 && idx >= 0 && idx < QUEUE_SIZE)
        return image_pool[idx];
    return nullptr;
};

int ImagePool::allocImage()
{
    int alloc = cur;
    cur = (cur + 1) % QUEUE_SIZE;
    return alloc;
}
